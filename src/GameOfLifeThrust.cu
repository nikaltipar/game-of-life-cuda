#include "GameOfLifeThrust.hpp"
#include <algorithm>
#include <random>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/discard_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/transform_output_iterator.h>
#include <thrust/iterator/zip_iterator.h>
#include <thrust/pair.h>
#include <thrust/reduce.h>

#include <cuda/std/mdspan>
#include <hip/hip_runtime.h>

__constant__ thrust::pair<int8_t, int8_t> neighbor_offsets[8] = {
    {-1, -1}, {0, -1}, {1, -1}, {-1, 0}, {1, 0}, {-1, 1}, {0, 1}, {1, 1}
};

GameOfLifeThrust::GameOfLifeThrust(uint32_t width, uint32_t height, uint32_t seed)
    : GameOfLife(width, height, seed)
    , grid(width * height)
    , next_grid(width * height)
{
}

void GameOfLifeThrust::print()
{
    for (uint32_t i = 0; i < height; i++)
    {
        for (uint32_t j = 0; j < width; j++)
        {
            std::cout << static_cast<uint32_t>(grid[i * width + j]) << " ";
        }
        std::cout << std::endl;
    }
}

void GameOfLifeThrust::initialize()
{
    std::mt19937 gen(seed);
    std::bernoulli_distribution dist(0.5f);

    for (uint32_t i = 0; i < width * height; i++)
    {
        grid[i] = dist(gen);
    }
}

void GameOfLifeThrust::play(uint32_t steps)
{
    cuda::std::mdspan grid_span(thrust::raw_pointer_cast(grid.data()), height, width);

    const uint32_t width = this->width;
    const uint32_t height = this->height;

    for (uint32_t step = 0; step < steps; step++)
    {
        auto count_iterator = thrust::make_counting_iterator<uint32_t>(0);
        auto cell_neighbour_iter = thrust::make_transform_iterator(count_iterator, [=] __host__ __device__(uint32_t i) {
            uint32_t cell_index = i / 8;
            uint32_t neighbor_index = i % 8;

            uint32_t cell_x = cell_index / width;
            uint32_t cell_y = cell_index % width;

            auto neighbor = neighbor_offsets[neighbor_index];
            int64_t nx = neighbor.first + cell_x;
            int64_t ny = neighbor.second + cell_y;

            return thrust::make_tuple(cell_x, cell_y, nx, ny);
        });

        auto cell_neighbour_values_iter = thrust::make_transform_iterator(
            cell_neighbour_iter,
            [=] __host__ __device__(thrust::tuple<uint32_t, uint32_t, int64_t, int64_t> t) {
                auto [cell_x, cell_y, nx, ny] = t;
                uint32_t i = cell_x * width + cell_y;
                uint32_t neighbor_value = (nx >= 0 && nx < height && ny >= 0 && ny < width) ? grid_span(nx, ny) : 0U;
                return thrust::make_pair(i, neighbor_value);
            }
        );

        auto cell_neighbour_get_cell_index = thrust::make_transform_iterator(
            cell_neighbour_values_iter, [=] __host__ __device__(thrust::pair<uint32_t, uint32_t> t) { return t.first; }
        );

        auto cell_neighbour_get_neigh_value = thrust::make_transform_iterator(
            cell_neighbour_values_iter, [=] __host__ __device__(thrust::pair<uint32_t, uint32_t> t) { return t.second; }
        );

        thrust::reduce_by_key(
            thrust::device,
            cell_neighbour_get_cell_index,
            cell_neighbour_get_cell_index + (width * height * 8),
            cell_neighbour_get_neigh_value,
            thrust::make_discard_iterator(),
            next_grid.begin()
        );

        auto input_sum_zip_iterator = thrust::make_zip_iterator(grid.begin(), next_grid.begin());
        thrust::transform(
            thrust::device,
            input_sum_zip_iterator,
            input_sum_zip_iterator + (width * height),
            next_grid.begin(),
            [=] __host__ __device__(thrust::tuple<Tile, uint8_t> t) {
                auto [val, sum] = t;
                return static_cast<Tile>((sum == 3) || (sum == 2 && val));
            }
        );

        thrust::swap(grid, next_grid);
    }
}

std::vector<Tile> GameOfLifeThrust::get_grid() const
{
    return std::vector<Tile>(grid.begin(), grid.end());
}